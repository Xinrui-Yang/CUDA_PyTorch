#include "hip/hip_runtime.h"

#include <stdio.h>

#define BLOCK_WIDTH 16

__global__ void MatrixMulKernel(float *A, float *B, float *C, int rows_A, int cols_A, int cols_B)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < rows_A && col < cols_B)
	{
		float value = 0;
		for (int i = 0; i < cols_A; i++)
		{
			value += A[row * cols_A + i] * B[i * cols_B + col];
		}
		C[row * cols_B + col] = value;
	}
}

void MatrixMul_cuda(float *A, float *B, float *C, int rows_A, int cols_A, int cols_B)
{
	float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, rows_A*cols_A*sizeof(float));
    hipMalloc((void**)&d_B, cols_A*cols_B*sizeof(float));
	hipMalloc((void**)&d_C, rows_A*cols_B*sizeof(float));

	hipMemcpy(d_A, A, rows_A*cols_A*sizeof(float), hipMemcpyHostToDevice);
  	hipMemcpy(d_B, B, cols_A*cols_B*sizeof(float), hipMemcpyHostToDevice);

	dim3 grid((cols_B + BLOCK_WIDTH - 1) / BLOCK_WIDTH, (rows_A + BLOCK_WIDTH - 1) / BLOCK_WIDTH);
	dim3 block(BLOCK_WIDTH, BLOCK_WIDTH);
	MatrixMulKernel<<<grid, block>>>(d_A, d_B, d_C, rows_A, cols_A, cols_B);

	hipMemcpy(C, d_C, rows_A*cols_B*sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}